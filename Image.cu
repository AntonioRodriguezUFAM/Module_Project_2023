#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include "Image.h"
#include <stdio.h>
#include "../include/stb_image.h"
#include "../include/stb_image_write.h"
#include <iostream>
#include <string>
#include <cassert>
#include <chrono>
using namespace std::chrono;

//#define STB_IMAGE_IMPLEMENTATION
// Write Images
//#define STB_IMAGE_WRITE_IMPLEMENTATION

//#define STBIR_INCLUDE_STB_IMAGE_RESIZE_H

//#include "include/stb_image.h"
//#include "include/stb_image_resize.h"
//#include "include/stb_image_write.h"
struct Pixel
{
	unsigned char r, g, b, a;
};

__global__ void AddKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
	ptrPixel->r = pixelValue;
	ptrPixel->g = pixelValue;
	ptrPixel->b = pixelValue;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToRedGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 1;
	ptrPixel->g = ptrPixel->g * 0.5;
	ptrPixel->b = ptrPixel->b * 0.5;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToGreenGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 0.5;
	ptrPixel->g = ptrPixel->g * 1;
	ptrPixel->b = ptrPixel->b * 0.5;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToBlueGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 0.5;
	ptrPixel->g = ptrPixel->g * 0.5;
	ptrPixel->b = ptrPixel->b * 1;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToInvGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * -1 + 255;
	ptrPixel->g = ptrPixel->g * -1 + 255;
	ptrPixel->b = ptrPixel->b * -1 + 255;
	ptrPixel->a = 255;
}


using namespace std;
void useGPU::adding(int* c, const int* a, const int* b, unsigned int size) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	AddKernel << <1, size >> > (dev_c, dev_a, dev_b);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

int useGPU::ImageToGrayGpu(unsigned char* imageRGBA, int width, int height) {
	// Cuda Event Timers
	float GPUTime = 0.0f;
	hipEvent_t start_01;
	hipEvent_t stop_01;
	hipEventCreate(&start_01);
	hipEventCreate(&stop_01);

	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	// Start Cuda Timer
	hipEventRecord(start_01);
	ConvertImageToGrayGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	// Stop Cuda Timer
	hipEventRecord(stop_01);
	//Compute Cuda Timer
	hipEventElapsedTime(&GPUTime, start_01, stop_01);

	cout << "GPU Timer: " << (int)GPUTime << endl;

	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	return duration.count();
	// Build output filename
	//std::string fileNameOut = "images/output.jpg";
	//stbi_write_jpg("images/output.jpg", width, height, 3, imageRGBA, 100);
}

int useGPU::ImageToRedGpu(unsigned char* imageRGBA, int width, int height) {
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToRedGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	return duration.count();
	// Build output filename
	//std::string fileNameOut = "images/output.jpg";
	//stbi_write_jpg("images/output.jpg", width, height, 3, imageRGBA, 100);
}

int useGPU::ImageToGreenGpu(unsigned char* imageRGBA, int width, int height) {
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToGreenGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	return duration.count();
	// Build output filename
	//std::string fileNameOut = "images/output.jpg";
	//stbi_write_jpg("images/output.jpg", width, height, 3, imageRGBA, 100);
}

int useGPU::ImageToBlueGpu(unsigned char* imageRGBA, int width, int height) {
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToBlueGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	return duration.count();
	// Build output filename
	//std::string fileNameOut = "images/output.jpg";
	//stbi_write_jpg("images/output.jpg", width, height, 3, imageRGBA, 100);
}

int useGPU::ImageToInvGpu(unsigned char* imageRGBA, int width, int height) {
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToInvGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	return duration.count();
	// Build output filename
	//std::string fileNameOut = "images/output.jpg";
	//stbi_write_jpg("images/output.jpg", width, height, 3, imageRGBA, 100);
}