#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <stdio.h>
#include "include/stb_image.h"
#include "include/stb_image_write.h"
#include <iostream>
#include <string>
#include <cassert>
#include <chrono>

// GPU Filters
#include"Filter_CPU.h"
#include "Filter_Gpu.h"



using namespace std::chrono;


// Create Data Structure Pixel
struct Pixel
{
	unsigned char r, g, b, a;
};

__global__ void AddKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
	ptrPixel->r = pixelValue;
	ptrPixel->g = pixelValue;
	ptrPixel->b = pixelValue;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToRedGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 1;
	ptrPixel->g = ptrPixel->g * 0.5;
	ptrPixel->b = ptrPixel->b * 0.5;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToGreenGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 0.5;
	ptrPixel->g = ptrPixel->g * 1;
	ptrPixel->b = ptrPixel->b * 0.5;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToBlueGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 0.5;
	ptrPixel->g = ptrPixel->g * 0.5;
	ptrPixel->b = ptrPixel->b * 1;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToInvGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * -1 + 255;
	ptrPixel->g = ptrPixel->g * -1 + 255;
	ptrPixel->b = ptrPixel->b * -1 + 255;
	ptrPixel->a = 255;
}

using namespace std;

void useGPU::adding(int* c, const int* a, const int* b, unsigned int size) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	AddKernel<<< 1, size >>> (dev_c, dev_a, dev_b);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}


int useGPU::ImageToGrayGpu(unsigned char* imageRGBA, int width, int height) {
	#define N 1000         // a big number up to INT_MAX, 2,147,483,647
	int T = 1, B = 1;            				// threads per block and blocks per grid
	float cpu_result, *gpu_result, ans[T * B];	// result from gpu, to make sure computation is being done

	// Start Timers
	hipEvent_t startgpu, endgpu;    				// using cuda events to measure time
	float time;       						// which is applicable for asynchronous code also
	hipEventCreate(&startgpu);    		 	// instrument code to measure start time
	hipEventCreate(&endgpu);

	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);



	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	// CUDA  Start reconding the Time
	hipEventRecord(startgpu, 0);

	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	// 
	// STOP  TIMER
	hipEventRecord(endgpu, 0);    	 		// instrument code to measure end time
	
	auto start = high_resolution_clock::now();
	ConvertImageToGrayGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);

	
	hipEventSynchronize(endgpu);	// wait for all device work to complete
	hipEventElapsedTime(&time, startgpu, endgpu);

	printf("GPU, Answer thread 0, %e\n", ans[0]);
	printf("GPU Number of floating pt operations done %e\n", (double)N * N * T * B);
	printf("GPU Time using CUDA events: %f ms\n", time);  		// time is in ms

	hipEventDestroy(startgpu); //destroy start event
	hipEventDestroy(endgpu);	//destroy stop event

	return duration.count();

	// Time of execution
	//return timeGpu;

	
}

double useGPU::ImageToRedGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();

	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start1 = high_resolution_clock::now();

	ConvertImageToRedGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);

	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	return timecpu;
}

double useGPU::ImageToGreenGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);

	ConvertImageToGreenGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);

	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	return timecpu;
}

double useGPU::ImageToBlueGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start1 = high_resolution_clock::now();
	ConvertImageToBlueGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	return timecpu;
}

double useGPU::ImageToInvGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start1 = high_resolution_clock::now();
	ConvertImageToInvGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	return timecpu;
}