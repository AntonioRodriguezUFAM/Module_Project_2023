#include "hip/hip_runtime.h"

// GPU Filters
#include"Filter_CPU.h"
#include "Filter_Gpu.h"

#include <stdio.h>
#include "include/stb_image.h"
#include "include/stb_image_write.h"
#include <iostream>
#include <string>
#include <cassert>
#include <chrono>
#include <ctime>


#include <msclr\marshal_cppstd.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""


using namespace std::chrono;
using namespace std;

// Create Data Structure Pixel
struct Pixel
{
	unsigned char r, g, b, a;
};

__global__ void AddKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
	ptrPixel->r = pixelValue;
	ptrPixel->g = pixelValue;
	ptrPixel->b = pixelValue;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToRedGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 1;
	ptrPixel->g = ptrPixel->g * 0.5;
	ptrPixel->b = ptrPixel->b * 0.5;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToGreenGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 0.5;
	ptrPixel->g = ptrPixel->g * 1;
	ptrPixel->b = ptrPixel->b * 0.5;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToBlueGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * 0.5;
	ptrPixel->g = ptrPixel->g * 0.5;
	ptrPixel->b = ptrPixel->b * 1;
	ptrPixel->a = 255;
}

__global__ void ConvertImageToInvGpu(unsigned char* imageRGBA)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t idx = y * blockDim.x * gridDim.x + x;

	Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
	unsigned char pixelValue = (unsigned char)
		//	(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
		ptrPixel->r = ptrPixel->r * -1 + 255;
	ptrPixel->g = ptrPixel->g * -1 + 255;
	ptrPixel->b = ptrPixel->b * -1 + 255;
	ptrPixel->a = 255;
}



void useGPU::adding(int* c, const int* a, const int* b, unsigned int size) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	AddKernel<<< 1, size >>> (dev_c, dev_a, dev_b);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}


int useGPU::ImageToGrayGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToGrayGpu << <gridSize, blockSize >> > (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	return duration.count();

	// Time of execution
	//return timeGpu;

	
}

double useGPU::ImageToRedGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();

	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();

	ConvertImageToRedGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);

	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	//return timecpu;
}

double useGPU::ImageToGreenGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);

	ConvertImageToGreenGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);

	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	//return timecpu;
}

double useGPU::ImageToBlueGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToBlueGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	//return timecpu;
}

double useGPU::ImageToInvGpu(unsigned char* imageRGBA, int width, int height) {
	// Start Timers
	auto start = std::chrono::steady_clock::now();
	unsigned char* ptrImageDataGpu = nullptr;
	assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
	assert(hipMemcpy(ptrImageDataGpu, imageRGBA, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);

	// Process image on gpu
	dim3 blockSize(20, 20);
	dim3 gridSize(width / blockSize.x, height / blockSize.y);
	//ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
	auto start = high_resolution_clock::now();
	ConvertImageToInvGpu <<< gridSize, blockSize >>> (ptrImageDataGpu);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);
	auto err = hipGetLastError();

	// Copy data from the gpu
	assert(hipMemcpy(imageRGBA, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);

	hipFree(ptrImageDataGpu);
	// Stop Timer
	auto end = std::chrono::steady_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	auto Duration_ms = duration_cast<microseconds> (elapsed_seconds);
	double timecpu = Duration_ms.count();

	// Time of execution
	//return timecpu;
}